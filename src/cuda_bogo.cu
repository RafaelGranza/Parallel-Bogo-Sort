
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

bool cuda_is_ordered(vector<int>& vec){
    return is_sorted(vec.begin(), vec.end());
}

void cuda_next(vector<int>& vec){
    random_shuffle(vec.begin(), vec.end());
}

vector<int> cuda_bogo(vector<int> vec){
    while(not cuda_is_ordered(vec)){
        cuda_next(vec);
    }
    return vec;
}